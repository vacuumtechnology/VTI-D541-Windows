#include "hip/hip_runtime.h"
#include <pcl/io/pcd_io.h>
#include <pcl/point_cloud.h>
#include <pcl/correspondence.h>
#include <pcl/features/normal_3d_omp.h>
#include <pcl/features/shot_omp.h>
#include <pcl/features/board.h>
#include <pcl/filters/uniform_sampling.h>
#include <pcl/recognition/cg/hough_3d.h>
#include <pcl/recognition/cg/geometric_consistency.h>
#include <pcl/visualization/pcl_visualizer.h>
#include <pcl/kdtree/kdtree_flann.h>
#include <pcl/kdtree/impl/kdtree_flann.hpp>
#include <pcl/common/transforms.h>
#include <pcl/console/parse.h>
#include <pcl/filters/statistical_outlier_removal.h>
#include <pcl/filters/filter.h>
#include <map>
#include <cmath>

typedef pcl::PointXYZRGBA PointType;
typedef pcl::Normal NormalType;
typedef pcl::ReferenceFrame RFType;
typedef pcl::SHOT352 DescriptorType;

class ObjectDetector {
public:
    ObjectDetector(pcl::PointCloud<PointType>::Ptr sceneCloud, pcl::PointCloud<PointType>::Ptr modelCloud, std::string configFile);
    float CalculateResolution();
    void RemoveOutliers(bool processScene, bool processModel);
    void LoadParams(float model_ss, float scene_ss, float descr_rad, float cg_size, float cg_thresh, float rf_rad);
    void Detect();

    void ComputeNormals(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals);
    void Downsample(pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints);
    void ComputeDescriptors(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals,
        pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints,
        pcl::PointCloud<DescriptorType>::Ptr model_descriptors, pcl::PointCloud<DescriptorType>::Ptr scene_descriptors);
    void FindCorrespondences(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals,
        pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints,
        pcl::PointCloud<DescriptorType>::Ptr model_descriptors, pcl::PointCloud<DescriptorType>::Ptr scene_descriptors);

    void DetermineBestMatches(int max_objects);
    void PrintInstances();
    void VisualizeResults();

private:
    pcl::PointCloud<PointType>::Ptr scene;
    pcl::PointCloud<PointType>::Ptr model;
    std::vector<Eigen::Matrix4f, Eigen::aligned_allocator<Eigen::Matrix4f> > rototranslations;
    std::vector<pcl::Correspondences> clustered_corrs;
    std::multimap<size_t, int> bestMatches;
    std::multimap<size_t, int>::reverse_iterator it;
    std::multimap<size_t, int>::reverse_iterator it2;

    float model_ss;
    float scene_ss;
    float descr_rad;
    float cg_size;
    float cg_thresh;
    float rf_rad;
    int max_objects;
    int min_distance;
    std::string config;
};

//
// Constructor
//
ObjectDetector::ObjectDetector(pcl::PointCloud<PointType>::Ptr sceneCloud, pcl::PointCloud<PointType>::Ptr modelCloud, std::string configFile) {
    this->scene = sceneCloud;
    this->model = modelCloud;
    this->config = configFile;
}

//
// Calculates resolution of model which is later used to adjust parameters
//
float ObjectDetector::CalculateResolution() {
    float resolution = 0.0;
    int n_points = 0;
    int nres;
    std::vector<int> indices(2);
    std::vector<float> sqr_distances(2);
    pcl::search::KdTree<PointType> tree;
    tree.setInputCloud(scene);

    std::cout << "Calculating Resolution" << endl;

    for (std::size_t i = 0; i < scene->size(); ++i)
    {
        if (!std::isfinite((*scene)[i].x))
        {
            continue;
        }
        //Considering the second neighbor since the first is the point itself.
        nres = tree.nearestKSearch(i, 2, indices, sqr_distances);
        if (nres == 2)
        {
            resolution += sqrt(sqr_distances[1]);
            ++n_points;
        }
    }
    if (n_points != 0)
    {
        resolution /= n_points;
    }

    return resolution;
}

//
// Removes Statistical Outliers to improve accuracy and decrease number of points to be processed
//
void ObjectDetector::RemoveOutliers(bool processScene, bool processModel) {
    pcl::StatisticalOutlierRemoval<pcl::PointXYZRGBA> sor;
    std::cout << "RemoveOutliers" << endl;

    if (processScene) {
        sor.setInputCloud(scene);
        sor.setMeanK(50);
        sor.setStddevMulThresh(.85);
        sor.filter(*scene);
    }

    if (processModel) {
        sor.setInputCloud(model);
        sor.setMeanK(50);
        sor.setStddevMulThresh(.85);
        sor.filter(*model);
    }
}

//
// Calculates resolution of model which is later used to adjust parameters
//
void ObjectDetector::LoadParams(float model_ss, float scene_ss, float descr_rad, float cg_size, float cg_thresh, float rf_rad) {
    float resolution = CalculateResolution();
    this->model_ss = model_ss * resolution;
    this->scene_ss = scene_ss * resolution;
    this->descr_rad = descr_rad * resolution;
    this->cg_size = cg_size * resolution;
    this->cg_thresh = cg_thresh;
    this->rf_rad = rf_rad * resolution;

    std::cout << "Model resolution:       " << resolution << std::endl;
    std::cout << "Model sampling size:    " << this->model_ss << std::endl;
    std::cout << "Scene sampling size:    " << this->scene_ss << std::endl;
    std::cout << "LRF support radius:     " << this->rf_rad << std::endl;
    std::cout << "SHOT descriptor radius: " << this->descr_rad << std::endl;
    std::cout << "Clustering bin size:    " << this->cg_size << std::endl << std::endl;
}

//
// Calls all member functions necessary for detection
//
void ObjectDetector::Detect() {
    pcl::PointCloud<NormalType>::Ptr model_normals(new pcl::PointCloud<NormalType>());
    pcl::PointCloud<NormalType>::Ptr scene_normals(new pcl::PointCloud<NormalType>());
    pcl::PointCloud<PointType>::Ptr model_keypoints(new pcl::PointCloud<PointType>());
    pcl::PointCloud<PointType>::Ptr scene_keypoints(new pcl::PointCloud<PointType>());
    pcl::PointCloud<DescriptorType>::Ptr model_descriptors(new pcl::PointCloud<DescriptorType>());
    pcl::PointCloud<DescriptorType>::Ptr scene_descriptors(new pcl::PointCloud<DescriptorType>());

    this->ComputeNormals(model_normals, scene_normals);
    this->Downsample(model_keypoints, scene_keypoints);
    this->ComputeDescriptors(model_normals, scene_normals, model_keypoints, scene_keypoints, model_descriptors, scene_descriptors);
    this->FindCorrespondences(model_normals, scene_normals, model_keypoints, scene_keypoints, model_descriptors, scene_descriptors);

}

//
// Computes normal vectors of all points in model and scene
//
void ObjectDetector::ComputeNormals(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals) {
    std::cout << "ComputeNormals" << endl;

    pcl::NormalEstimationOMP<PointType, NormalType> norm_est;
    norm_est.setKSearch(15);
    norm_est.setInputCloud(model);
    norm_est.compute(*model_normals);
    norm_est.setInputCloud(scene);
    norm_est.compute(*scene_normals);

}

//
//  Downsample Clouds to Extract keypoints
//
void ObjectDetector::Downsample(pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints) {
    std::cout << "Downsample" << endl;

    pcl::UniformSampling<PointType> uniform_sampling;
    uniform_sampling.setInputCloud(model);
    uniform_sampling.setRadiusSearch(model_ss);
    uniform_sampling.filter(*model_keypoints);
    std::cout << "Model total points: " << model->size() << "; Selected Keypoints: " << model_keypoints->size() << std::endl;

    uniform_sampling.setInputCloud(scene);
    uniform_sampling.setRadiusSearch(scene_ss);
    uniform_sampling.filter(*scene_keypoints);
    std::cout << "Scene total points: " << scene->size() << "; Selected Keypoints: " << scene_keypoints->size() << std::endl;

}


//
//  Compute Descriptor for keypoints
//
void ObjectDetector::ComputeDescriptors(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals,
    pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints,
    pcl::PointCloud<DescriptorType>::Ptr model_descriptors, pcl::PointCloud<DescriptorType>::Ptr scene_descriptors) {
    std::cout << "ComputeDescriptors" << endl;

    pcl::SHOTEstimationOMP<PointType, NormalType, DescriptorType> descr_est;
    descr_est.setRadiusSearch(descr_rad);

    descr_est.setInputCloud(model_keypoints);
    descr_est.setInputNormals(model_normals);
    descr_est.setSearchSurface(model);
    descr_est.compute(*model_descriptors);

    descr_est.setInputCloud(scene_keypoints);
    descr_est.setInputNormals(scene_normals);
    descr_est.setSearchSurface(scene);
    descr_est.compute(*scene_descriptors);
}

void ObjectDetector::FindCorrespondences(pcl::PointCloud<NormalType>::Ptr model_normals, pcl::PointCloud<NormalType>::Ptr scene_normals,
    pcl::PointCloud<PointType>::Ptr model_keypoints, pcl::PointCloud<PointType>::Ptr scene_keypoints,
    pcl::PointCloud<DescriptorType>::Ptr model_descriptors, pcl::PointCloud<DescriptorType>::Ptr scene_descriptors) {

    std::cout << "FindCorrespondences" << endl;

    //
    //  Find Model-Scene Correspondences with KdTree
    //
    pcl::CorrespondencesPtr model_scene_corrs(new pcl::Correspondences());

    pcl::KdTreeFLANN<DescriptorType> match_search;
    match_search.setInputCloud(model_descriptors);

    //  For each scene descriptor, find nearest neighbor into the model descriptor cloud and add it to the correspondences vector.
    for (std::size_t i = 0; i < scene_descriptors->size(); ++i)
    {
        std::vector<int> neigh_indices(1);
        std::vector<float> neigh_sqr_dists(1);
        if (!std::isfinite(scene_descriptors->at(i).descriptor[0])) //skipping NaNs
        {
            continue;
        }
        cout << "";
        int found_neighs = match_search.nearestKSearch(scene_descriptors->at(i), 1, neigh_indices, neigh_sqr_dists);
        if (found_neighs == 1 && neigh_sqr_dists[0] < 0.25f) //  add match only if the squared descriptor distance is less than 0.25 (SHOT descriptor distances are between 0 and 1 by design)
        {
            pcl::Correspondence corr(neigh_indices[0], static_cast<int> (i), neigh_sqr_dists[0]);
            model_scene_corrs->push_back(corr);
        }
    }
    std::cout << "Correspondences found: " << model_scene_corrs->size() << std::endl;

    //
    //  Compute (Keypoints) Reference Frames for Hough
    //
    pcl::PointCloud<RFType>::Ptr model_rf(new pcl::PointCloud<RFType>());
    pcl::PointCloud<RFType>::Ptr scene_rf(new pcl::PointCloud<RFType>());

    pcl::BOARDLocalReferenceFrameEstimation<PointType, NormalType, RFType> rf_est;
    rf_est.setFindHoles(false);
    rf_est.setRadiusSearch(rf_rad);

    rf_est.setInputCloud(model_keypoints);
    rf_est.setInputNormals(model_normals);
    rf_est.setSearchSurface(model);
    rf_est.compute(*model_rf);

    rf_est.setInputCloud(scene_keypoints);
    rf_est.setInputNormals(scene_normals);
    rf_est.setSearchSurface(scene);
    rf_est.compute(*scene_rf);

    //  Clustering
    pcl::Hough3DGrouping<PointType, PointType, RFType, RFType> clusterer;
    clusterer.setHoughBinSize(cg_size);
    clusterer.setHoughThreshold(cg_thresh);
    clusterer.setUseInterpolation(true);
    clusterer.setUseDistanceWeight(false);

    clusterer.setInputCloud(model_keypoints);
    clusterer.setInputRf(model_rf);
    clusterer.setSceneCloud(scene_keypoints);
    clusterer.setSceneRf(scene_rf);
    clusterer.setModelSceneCorrespondences(model_scene_corrs);

    clusterer.recognize(rototranslations, clustered_corrs);

    std::cout << "Correspondences Found" << endl;

}

void ObjectDetector::DetermineBestMatches(int max_objects) {

    // Sort matches by # of correspondences
    for (std::size_t i = 0; i < rototranslations.size(); ++i) {
        bestMatches.insert(std::make_pair(clustered_corrs[i].size(), i));
    }

    int i = 0;
    int j = 0;
    float xdiff, ydiff, zdiff, dist;
    Eigen::Vector3f translation, translation2;

    // Detects duplicate detections
    for (it = bestMatches.rbegin(); it != bestMatches.rend(); it++) {
        translation = rototranslations[it->second].block<3, 1>(0, 3);
        for (it2 = it; it2 != bestMatches.rend(); it2++) {
            if (j >= max_objects) break;
            translation2 = rototranslations[it2->second].block<3, 1>(0, 3);

            // Calculate distance between two translations
            xdiff = translation2.x() - translation.x();
            ydiff = translation2.y() - translation.y();
            zdiff = translation2.z() - translation.z();


            /*std::cout << "xdiff " << xdiff;
            std::cout << ", ydiff " << ydiff;
            std::cout << ", zdiff " << zdiff;*/

            //dist = sqrt(pow(xdiff, 2) + pow(ydiff, 2) + pow(zdiff, 2));
            dist = sqrt(pow(xdiff, 2) + pow(ydiff, 2));
            //std::cout << ", Instances " << i << " " << j << ". Distance: " << dist << endl;
            j++;
        }
        i++;
        j = i;
        if (i >= max_objects) break;
    }
    this->max_objects = max_objects;
}

//
//  Output results
//
void ObjectDetector::PrintInstances() {

    int c = 0;
    for (it = bestMatches.rbegin(); it != bestMatches.rend(); it++) {
        std::cout << "\n    Instance " << c << ":" << std::endl;
        std::cout << "        Correspondences belonging to this instance: " << clustered_corrs[it->second].size() << std::endl;

        // Print the rotation matrix and translation vector
        Eigen::Matrix3f rotation = rototranslations[it->second].block<3, 3>(0, 0);
        Eigen::Vector3f translation = rototranslations[it->second].block<3, 1>(0, 3);

        printf("\n");
        printf("            | %6.3f %6.3f %6.3f | \n", rotation(0, 0), rotation(0, 1), rotation(0, 2));
        printf("        R = | %6.3f %6.3f %6.3f | \n", rotation(1, 0), rotation(1, 1), rotation(1, 2));
        printf("            | %6.3f %6.3f %6.3f | \n", rotation(2, 0), rotation(2, 1), rotation(2, 2));
        printf("\n");
        printf("        t = < %0.3f, %0.3f, %0.3f >\n", translation(0), translation(1), translation(2));
        c++;
        if (c >= max_objects) break;
    }
}

//
//  Visualization
//
void ObjectDetector::VisualizeResults() {

    pcl::visualization::PCLVisualizer viewer("Correspondence Grouping");
    viewer.addPointCloud(scene, "scene_cloud");

    pcl::PointCloud<PointType>::Ptr off_scene_model(new pcl::PointCloud<PointType>());
    pcl::PointCloud<PointType>::Ptr off_scene_model_keypoints(new pcl::PointCloud<PointType>());

    int c = 0;
    for (it = bestMatches.rbegin(); it != bestMatches.rend(); it++) {
        pcl::PointCloud<PointType>::Ptr rotated_model(new pcl::PointCloud<PointType>());
        pcl::transformPointCloud(*model, *rotated_model, rototranslations[it->second]);

        std::stringstream ss_cloud;
        ss_cloud << "instance" << c;

        pcl::visualization::PointCloudColorHandlerCustom<PointType> rotated_model_color_handler(rotated_model, 255, 0, 0);
        viewer.addPointCloud(rotated_model, rotated_model_color_handler, ss_cloud.str());

        c++;
        if (c >= max_objects) break;
    }

    while (!viewer.wasStopped())
    {
        viewer.spinOnce();
    }
}


void
showHelp(char* filename)
{
    std::cout << endl;
    std::cout << "Usage: " << filename << " model_filename.pcd scene_filename.pcd config_filename.txt" << std::endl << std::endl;
    std::cout << "Configuration:" << std::endl;
    std::cout << "     model_ss = val         Model uniform sampling radius" << std::endl;
    std::cout << "     scene_ss = val         Scene uniform sampling radius" << std::endl;
    std::cout << "     rf_rad = val           Reference frame radius" << std::endl;
    std::cout << "     descr_rad = val        Descriptor radius" << std::endl;
    std::cout << "     cg_size = val          Cluster size" << std::endl;
    std::cout << "     max_objects = val      Number of objects to detect" << std::endl;
    std::cout << "     cg_thresh = val        Clustering threshold" << std::endl << std::endl;
}

int main(int argc, char** argv) {
    pcl::PointCloud<PointType>::Ptr model(new pcl::PointCloud<PointType>());
    pcl::PointCloud<PointType>::Ptr scene(new pcl::PointCloud<PointType>());
    float model_ss;
    float scene_ss;
    float rf_rad;
    float descr_rad;
    float cg_size;
    float cg_thresh;
    int max_objects;

    std::string modelFile = argv[1];
    std::string sceneFile = argv[2];
    std::string configFile = argv[3];

    //  Load clouds
    if (pcl::io::loadPCDFile(modelFile, *model) < 0) {
        std::cout << "Error loading model cloud." << std::endl;
        showHelp(argv[0]);
        return (-1);
    }
    if (pcl::io::loadPCDFile(sceneFile, *scene) < 0) {
        std::cout << "Error loading scene cloud." << std::endl;
        showHelp(argv[0]);
        return (-1);
    }

    // Load config
    std::ifstream cFile(configFile);
    if (cFile.is_open())
    {
        std::string line;
        while (getline(cFile, line)) {
            line.erase(std::remove_if(line.begin(), line.end(), isspace),
                line.end());
            if (line[0] == '#' || line.empty())
                continue;
            auto delimiterPos = line.find("=");
            auto name = line.substr(0, delimiterPos);
            auto value = line.substr(delimiterPos + 1);

            if (name == "model_ss") model_ss = atof(value.c_str());
            else if (name == "scene_ss") scene_ss = atof(value.c_str());
            else if (name == "rf_rad") rf_rad = atof(value.c_str());
            else if (name == "descr_rad") descr_rad = atof(value.c_str());
            else if (name == "cg_size") cg_size = atof(value.c_str());
            else if (name == "cg_thresh") cg_thresh = atof(value.c_str());
            else if (name == "max_objects") max_objects = atoi(value.c_str());

        }

    } else {
        std::cerr << "Couldn't open config file for reading.\n";
    }

    ObjectDetector* obj = new ObjectDetector(scene, model, configFile);

    obj->RemoveOutliers(true, true);

    obj->LoadParams(model_ss, scene_ss, descr_rad, cg_size, cg_thresh, rf_rad);
    obj->Detect();

    obj->DetermineBestMatches(max_objects);

    obj->PrintInstances();
    obj->VisualizeResults();

    delete(obj);
    return 0;
}